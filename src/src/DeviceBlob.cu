//
// Created by Adam Kosiorek on 6/12/15.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include "DeviceBlob.h"
#include "CudaUtils.h"

template<class Dtype>
DeviceBlob<Dtype>::DeviceBlob() : rows_(0), cols_(0), count_(0), bytes_(0), data_(NULL) {}

template<class Dtype>
DeviceBlob<Dtype>::DeviceBlob(int rows, int cols)
        : rows_(rows), cols_(cols), count_(rows_ * cols_), bytes_(count_ * sizeof(Dtype)) {
    CUDA_CHECK(hipMalloc(&data_, bytes_));
}

template<class Dtype>
DeviceBlob<Dtype>::DeviceBlob(int rows, int cols, const Dtype* from) : DeviceBlob(rows, cols) {
    copyFrom(from);
}

template<class Dtype>
DeviceBlob<Dtype>::DeviceBlob(const DeviceBlob& that) : DeviceBlob(that.rows(), that.cols()) {
    if(that.data() != NULL && bytes_ != 0) {
        CUDA_CHECK(hipMemcpy(data_, that.data(), bytes_, hipMemcpyDeviceToDevice));
    }
}

template<class Dtype>
DeviceBlob<Dtype>::~DeviceBlob() {
    hipFree(data_);
    data_ = NULL;
}

template<class Dtype>
DeviceBlob<Dtype>& DeviceBlob<Dtype>::operator= (DeviceBlob that) {
    swap(*this, that);
    return *this;
}


template<class Dtype>
void DeviceBlob<Dtype>::copyFrom(const Dtype* from) {
    CUDA_CHECK(hipMemcpy(data_, from, bytes_, hipMemcpyHostToDevice));
}

template<class Dtype>
void DeviceBlob<Dtype>::copyTo(Dtype* to) const {
    CUDA_CHECK(hipMemcpy(to, data_, bytes_, hipMemcpyDeviceToHost));
}

template<class Dtype>
void DeviceBlob<Dtype>::setZero() {
    CUDA_CHECK(hipMemset(data_, 0, bytes_));
}

template<class Dtype>
size_t DeviceBlob<Dtype>::rows() const {
    return rows_;
}

template<class Dtype>
size_t DeviceBlob<Dtype>::cols() const {
    return cols_;
}

template<class Dtype>
size_t DeviceBlob<Dtype>::count() const {
    return count_;
}

template<class Dtype>
Dtype* DeviceBlob<Dtype>::data() {
    return data_;
}

template<class Dtype>
const Dtype* DeviceBlob<Dtype>::data() const{
    return data_;
}

template class DeviceBlob<float>;
template class DeviceBlob<thrust::complex<float> >;